#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <unistd.h>

#include "filterise.h"

#define BLOCKS 5040

#define THREADS_PER_BLOCK 960

__global__ void filterise(int * filter,unsigned char * image_in_d ,unsigned char * image_out_d){
	int x=0 ;
    	int y=0 ;
    	int new_x =0;
    	int new_y =0;
    	int fx=0 ,fy=0 ;
    	int sum=0 ;			
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;     
	
	x= blockIdx.x * blockDim.x;
	y= threadIdx.x;	

	if(i<IMAGE_SIZE){
		
		sum = 0 ;
		for(fx=-1;fx<=1;fx++)
		{
			if( blockIdx.x-fx < 0 ) {  
				//block 0
				new_x = 5038 * blockDim.x ;			
			}
			else if( blockIdx.x-fx == 0 ) {  
				//block 1			
				new_x = 5039 * blockDim.x ;
			}
			else if(blockIdx.x-fx == 5039) { 
				new_x = 0 * blockDim.x ; 
			}
			else if(blockIdx.x-fx == 5040) { 
				new_x = 1 * blockDim.x ; 
			}
			else {
				new_x = x -fx* blockDim.x*2 ;
			}
			
			for(fy=-1;fy<=1;fy++)
			{    
				if(y-fy < 0){
					if(blockIdx.x%2 == 0){
						new_y = 959;
						new_x += blockDim.x;					
					}
					if(blockIdx.x%2 == 1){
						new_y = y-fy;					
					} 				
				}
				else if(y-fy == 960) {
					if(blockIdx.x%2 == 0){
						new_y = y-fy;					
					}
					if(blockIdx.x%2 == 1){
						new_y = 0 ;
						new_x -=blockDim.x ;					
					}
				}
				else{
					new_y = y -fy ;
				}

				sum += image_in_d[new_x+new_y]*filter[(fx+1)*3+(fy+1)];
			}
		}
		image_out_d[x + y] = (int)sum/16 ;
	}
	__syncthreads();		
}



void filterise_wrapper(int * filter,unsigned char * image_in_d ,unsigned char * image_out_d,unsigned char * image_in ,unsigned char * image_out){
        int i ;
        unsigned char * temp = NULL;
	
    
	

        //dim3 dimBl(BLOCK_SIZE);  
	//dim3 dimGr(HEIGHT); 
        
        for(i=0;i<1000;i++){
	    filterise<<< BLOCKS , THREADS_PER_BLOCK>>>(filter,image_in_d ,image_out_d );
	
	    temp = image_in_d ;
            image_in_d = image_out_d ;
            image_out_d = temp ;

	}
	
}
