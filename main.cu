#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <unistd.h>

#include "filterise.h"
#include "my_functions.h"
//#include "reduce.h"

int main(int argc, char** argv) {
    
    int i=0 ;
    //int keep_going =1;
    unsigned char k ;
    unsigned char * image_in = NULL ;
    unsigned char * image_out = NULL ;
    int * cpu_filter = NULL ;
    FILE * oldfile = NULL;
    FILE * newfile = NULL;
    
    //Arrays for Cuda
    unsigned char * image_in_d = NULL ;
    unsigned char * image_out_d = NULL ;
    int * gpu_filter = NULL ;
    
    time_t start_t, end_t;
    double diff_t;
    
    oldfile = fopen("waterfall_grey_1920_2520.raw","r");
    newfile = fopen("out.raw","w");
    
    if(oldfile == NULL){
        printf("ERROR @ fopen : oldfile\n");
        return(EXIT_FAILURE);
    }
    
    if(newfile == NULL){
        printf("ERROR @ fopen : newfile\n");
        return(EXIT_FAILURE);
    }
    
    /*allocate pixel arrays*/
    image_in =(unsigned char *)my_malloc(IMAGE_SIZE * sizeof(unsigned char)); 
    image_out =(unsigned char *)my_malloc(IMAGE_SIZE * sizeof(unsigned char));
    
    /*Create filter*/
    cpu_filter = (int *)my_malloc(9*sizeof(int));
    cpu_filter[0]= 1;
    cpu_filter[1]= 2;
    cpu_filter[2]= 1;
    cpu_filter[3]= 2;
    cpu_filter[4]= 4;
    cpu_filter[5]= 2;
    cpu_filter[6]= 1;
    cpu_filter[7]= 2;
    cpu_filter[8]= 1;
    
    
    /*allocate memory Cuda...*/
    hipMalloc((void **)&image_in_d, IMAGE_SIZE * sizeof(unsigned char) );
    hipMalloc((void **)&image_out_d, IMAGE_SIZE * sizeof(unsigned char) );
    hipMalloc((void **)&gpu_filter, 9 * sizeof(int) );
    
    /*initialise pixel arrays*/
    for(i =0;i<IMAGE_SIZE;i++){
        image_in[i]  = 0;
        image_out[i] = 0;
    }
    
    /*passing from file to pixel array*/
    i=0;
    
    while( fread(&k,1,1,oldfile)>0){     
        image_in[i] = k;
        i++;
    }
    
    /*Copy arrays to GPU*/
    hipMemcpy(image_in_d, image_in, IMAGE_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice) ;
    hipMemset(image_out_d, 255, IMAGE_SIZE * sizeof(unsigned char)) ;
    hipMemcpy(gpu_filter, cpu_filter, 9 * sizeof(int), hipMemcpyHostToDevice) ;
    
    

    /*filterize*/
    time(&start_t);
    
    
        
	filterise_wrapper(gpu_filter,image_in_d ,image_out_d,image_in,image_out);
	/*keep_going = check_convergence(image_in,image_out);
	/*if(!keep_going)
	{
	    printf("Epanalipseis  = %d \n",w);
	    break;
	}*/
    
    
    time(&end_t);
    
    /*write to new file*/
    hipMemcpy(image_in, image_in_d, IMAGE_SIZE * sizeof(unsigned char), hipMemcpyDeviceToHost) ;
    fwrite(image_in,sizeof(unsigned char),IMAGE_SIZE,newfile);
    
    diff_t = difftime(end_t, start_t);
    printf("Execution time = %f seconds\n", diff_t);
    
    /*final actions*/
    
    fclose (oldfile);
    fclose (newfile);
    
    free(image_in);
    free(image_out);

    hipFree(image_in_d);
    hipFree(image_out_d);
    hipFree(gpu_filter);
    
    
    return (EXIT_SUCCESS);
}
